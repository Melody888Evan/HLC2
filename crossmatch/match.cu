#include "hip/hip_runtime.h"
#include <typeinfo>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include "manager.h"
#include <hipsparse.h>

unordered_map<int,vector<vector<double>>> recordmapAA;
unordered_map<int,vector<vector<double>>> recordmapBB;

__device__ double compute_distance(double rax,double decx,double ray, double decy){
    double tmp1 = rax - ray;
    double tmp2 = decx - decy;
    double tmp3 = decx + decy;
    tmp2 = tmp2*tmp2;
    tmp3 = cos(tmp3/360.0*PI); // try to define 1/360 as macro
    tmp1 = tmp1 * tmp3;
    tmp1 = tmp1 * tmp1;
    tmp1 = tmp1 + tmp2; 
    return tmp1;
}


//kernel function
__global__ void compute_1D_1D(double* d_in_ra1, double* d_in_dec1, double* d_in_ra2, double* d_in_dec2, int* d_out_dis, unsigned int nx, unsigned int ny,unsigned int *resultcount){

    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;	
    if (ix+1 > nx) return;	
	double d1 = d_in_ra1[ix], d2 = d_in_dec1[ix];
	for (unsigned int iy = 0; iy < ny; ++iy){
	    double tmp1 = compute_distance(d1,d2,d_in_ra2[iy], d_in_dec2[iy]);
	    if (tmp1<DIS){
	     d_out_dis[(iy)*(nx)+ix] = 1;
	     atomicAdd(resultcount, 1);
             }
	}
  
  //printf("max index: %d\n",maxindex);
	return;
}


int main(int argc, char **argv){

    double AllStart = cpuSecond();
    double iStart, iElaps;
  
    vector<double> parameters = parameter_decided();
    const int N = parameters[0];
    const int BLOCK_MAX_X = parameters[1];
    const int BLOCK_MAX_Y = parameters[2];
  
    //read the csv input into recordmap
    recordmapAA = read_to_unordered("data/twomasstest5.csv",recordmapAA);
    iElaps = cpuSecond() - AllStart;
    printf("A read to unorded time is %f s\n", iElaps);
    recordmapBB = read_to_unordered("data/twomasstest5.csv",recordmapBB);
    iElaps = cpuSecond() - AllStart;
    printf("A+B read to unorded time is %f s\n", iElaps);
    unordered_map<int,vector<vector<double>>>::iterator iterA;
    unordered_map<int,vector<vector<double>>>::iterator iterB;
    int linenumA,linenumB;
  	
   
    //get shared index list of recordmap A and B
    vector<int> sharedlist;
    sharedlist = get_shared_id(recordmapAA,recordmapBB);
    printf("shared index num: %d \n",sharedlist.size());
    iElaps = cpuSecond() - AllStart;
    printf("read csv file time is %f s\n", iElaps);
    //printf ("__________\n");
 
    vector<vector<double>> matchresult;
   
    // GPU Initialization
    // set up device
    int dev = 0;
    CHECK(hipSetDevice(dev));
   
   
   // define variety for communication of CPU and GPU
     double *h_in_ra1, *h_in_dec1, *h_in_ra2, *h_in_dec2;
     double *h_out_ra1, *h_out_dec1, *h_out_ra2, *h_out_dec2;
   
     double *d_in_ra1, *d_in_dec1, *d_in_ra2, *d_in_dec2;
     double *d_out_ra1, *d_out_dec1, *d_out_ra2, *d_out_dec2;
    
     double *h_in_ra_x, *h_in_dec_x, *h_in_ra_y, *h_in_dec_y;
     int *h_out_dis,*d_out_dis;
     unsigned int *h_count, *d_count;

     CHECK(hipHostMalloc((int**)&h_count, sizeof(int)));
     CHECK(hipMalloc((void **)&d_count, sizeof(int)));
     CHECK(hipMemset(d_count, 0, sizeof(int)));
  
     // malloc device memory space
     CHECK(hipHostMalloc((double**)&h_in_ra1, N*sizeof(double)));
     CHECK(hipHostMalloc((double**)&h_in_ra2, N*sizeof(double)));
     CHECK(hipHostMalloc((double**)&h_in_dec1, N*sizeof(double)));
     CHECK(hipHostMalloc((double**)&h_in_dec2, N*sizeof(double)));
   
     CHECK(hipHostMalloc((int**)&h_out_dis, BLOCK_MAX_X*BLOCK_MAX_Y*sizeof(int)));
     CHECK(hipMalloc((void **) &d_out_dis, BLOCK_MAX_X*BLOCK_MAX_Y*sizeof(int)));
    
    size_t nBytes, nBytes2, nBytes3;
    unsigned int resultcount = 0;
    unsigned int compress_resultcount = 0;
    unsigned int nx, ny;
    
    int streamcount=0;

     for(int i=0; i<sharedlist.size(); i++){
        vector<vector<double>> valuesA = recordmapAA[sharedlist[i]];
        vector<vector<double>> valuesB = recordmapBB[sharedlist[i]];
        for(int r=0; r<valuesA.size(); r++){
          h_in_ra1[r] = valuesA[r][2];
          h_in_dec1[r] = valuesA[r][3];
        }
        
        for(int t=0; t<valuesB.size(); t++){
          h_in_ra2[t] = valuesB[t][2];
          h_in_dec2[t] = valuesB[t][3];
        }
      
        // make it convenient to use varieties
    	if (valuesA.size() > valuesB.size()){
    		h_in_ra_x = h_in_ra1;
    		h_in_dec_x = h_in_dec1;
    		h_in_ra_y = h_in_ra2;
    		h_in_dec_y = h_in_dec2;
    		nx = valuesA.size();
    		ny = valuesB.size();
       	 }else{		
		h_in_ra_x = h_in_ra2;
    		h_in_dec_x = h_in_dec2;
    		h_in_ra_y = h_in_ra1;
    		h_in_dec_y = h_in_dec1;
    		nx = valuesB.size();
    		ny = valuesA.size();
    	}
       
      
      // for each calculation block
    	for (unsigned int data_x_offset = 0; data_x_offset < nx; data_x_offset += BLOCK_MAX_X){
    		for (unsigned int data_y_offset = 0; data_y_offset < ny; data_y_offset += BLOCK_MAX_Y){
            		streamcount = streamcount + 1; 
            
    			unsigned int data_x_band = min(nx-data_x_offset, BLOCK_MAX_X), 
    			data_y_band = min(ny-data_y_offset, BLOCK_MAX_Y);
    				
			unsigned int *h_sharedInteger,*d_sharedInteger;
			CHECK(hipHostMalloc((int**)&h_sharedInteger, sizeof(int)));
			CHECK(hipMalloc((void **)&d_sharedInteger, sizeof(int)));
			CHECK(hipMemset(d_sharedInteger, 0, sizeof(int)));
            
    			// dynamically set up the size of data
    			nBytes = (data_x_band)*sizeof(double);
    			nBytes2 = (data_y_band)*sizeof(double);
    			nBytes3 = (data_x_band)*(data_y_band)*sizeof(int);
    				
    			int dimx = 128;
    			int dimy = 1;
    
    			if (argc > 1) dimx = atoi(argv[1]);
    			dim3 block(dimx, dimy);
    			dim3 grid((data_x_band + block.x - 1) / block.x, 1);
    				
    			// execute the kernel and receive the result of GPU
    			iStart = cpuSecond();
    			CHECK(hipMalloc((void **) &d_in_ra1, nBytes));
    			CHECK(hipMalloc((void **) &d_in_dec1, nBytes));
    			CHECK(hipMalloc((void **) &d_in_ra2, nBytes2));
    			CHECK(hipMalloc((void **) &d_in_dec2,nBytes2));
    			//CHECK(hipMalloc((void **) &d_out_dis, nBytes3));
    
          		 //int j = streamcount % 2;                                  
    			CHECK(hipMemcpyAsync(d_in_ra1, h_in_ra_x+data_x_offset, nBytes, hipMemcpyHostToDevice));
    			CHECK(hipMemcpyAsync(d_in_dec1, h_in_dec_x+data_x_offset, nBytes, hipMemcpyHostToDevice));
    			CHECK(hipMemcpyAsync(d_in_ra2, h_in_ra_y+data_y_offset, nBytes2, hipMemcpyHostToDevice));
    			CHECK(hipMemcpyAsync(d_in_dec2, h_in_dec_y+data_y_offset,nBytes2, hipMemcpyHostToDevice));	
             		
            		compute_1D_1D <<<grid, block>>>(d_in_ra1, d_in_dec1, d_in_ra2, d_in_dec2, d_out_dis, data_x_band, data_y_band,d_sharedInteger);
            
    			CHECK(hipMemcpyAsync(h_out_dis, d_out_dis, nBytes3, hipMemcpyDeviceToHost));
            
    			 CHECK(hipMemcpy(h_sharedInteger, d_sharedInteger, sizeof(int),hipMemcpyDeviceToHost));
 
			//cout<<*h_sharedInteger<<endl;         
			resultcount= resultcount + *h_sharedInteger; 
  
    			CHECK(hipFree(d_in_ra1));
    			CHECK(hipFree(d_in_ra2));
    			CHECK(hipFree(d_in_dec1));
    			CHECK(hipFree(d_in_dec2));
			CHECK(hipFree(d_sharedInteger));
			CHECK(hipHostFree(h_sharedInteger));

    			}
    		}
       
       
    		iElaps = cpuSecond() - iStart;
    		//printf("[Info]File %d is done, elapsed %f s.\n", sharedlist[i], iElaps);
  	}
     

	 //check the correctness
	 printf("!!!!!!!--%ld--\n",resultcount);  
	 printf("[result:] %d \n",matchresult.size());
   
    
	//destroy memory
  	CHECK(hipHostFree(h_in_dec1));
  	CHECK(hipHostFree(h_in_dec2));
  	CHECK(hipHostFree(h_in_ra1));
  	CHECK(hipHostFree(h_in_ra2));
  	CHECK(hipHostFree(h_out_dis));
	CHECK(hipFree(d_out_dis)); 
	CHECK(hipFree(d_count));
	CHECK(hipHostFree(h_count));  
    

  	iElaps = cpuSecond() - AllStart;
  	printf("[Info]All time is %f s\n", iElaps);	
   
  	return 0;
  }
  
